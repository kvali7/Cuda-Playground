#include "hip/hip_runtime.h"

/* thanks old college classmate Sean Anderson!
 * http://graphics.stanford.edu/~seander/bithacks.html#DetermineIfPowerOf2
 */

//  #include <math.h>

//  #define BOARDSIZE 6
//  #define NUMQUEENS 3
 
 #define BOARDSIZE 6
 #define NUMQUEENS 3

 #include "checker_helper.cu"
#include "comb_creator.cu"
 
  __global__
  void qgdKernel(int n, int a, bool one, bool all, int pitch,
                 unsigned int * d_solutions, unsigned int * count, unsigned long int comb) {
      // this kernel is completely hardcoded to the 4x4 board
      // I'm not pretending otherwise
  
      int tid = threadIdx.x + blockIdx.x * blockDim.x;

    if ( all == true  ){
        // we know that for n=4, a=2, so knock out all boards where a != 2
        // int bitcount = countBits(tid);
        // if (bitcount != a) { return; }
    
        if (tid >= comb) return;

        int numQueens = NUMQUEENS;
        int width = BOARDSIZE;

        // create queens List for n = 4 the size of the proposed solution is 2
        unsigned int queensList[NUMQUEENS] = {0};
        // int temp = tid;
        // for (unsigned int c = 0, qi = 0 ; temp ; temp >>= 1, c++){
        //     if (temp & 1){
        //         queensList[qi] = c;
        //         qi++;
        //     }
        // }
        
        queenGen (queensList, tid, width * width, numQueens);
        if (checkerFunc (queensList, width, numQueens)) 
            addSolution (queensList,  numQueens, d_solutions, count, pitch);
    }
  }


  // Store your solutions in d_solution, which has already been allocated for you
  void qgd(int n, int a, bool one, bool all, int pitch,
           unsigned long long numSolutions, unsigned int * d_solutions,
           unsigned int * count) {
    
      // there are 2^16 possible configurations of queens on a 4x4 chessboard
      // 2^8 blocks of 2^8 threads each will check them all (brute force)
      int width = n;
      int numQueens = a;

      //all combinations
      printf("fact ");
      unsigned long int comb = 1;
      int m = width*width;
      int k = numQueens;
      for (int j = m ; j > m - k; j--){
            comb = comb * j;
      }
      for (int q = k; q > 0; q--){
          comb = comb / q;
      }
      printf("The number of the total combinations is %u\n", comb);
      //generate every possible combinations on the memory
    

      qgdKernel<<< 1<<8, 1<<8 >>>(width, numQueens, one, all, pitch, d_solutions, count, comb);
    //   qgdKernel<<< 1<<10, 1<<6 >>>(n, a, one, all, pitch, d_solutions, count)
    int even = !(width % 2);
    printf("Is it even? %d\n", even);
    // printf("number of found solutions = %u\n", *count);

    // unsigned int* solution = (unsigned int*) ((char*) d_solutions + solution_id * pitch);


    // unsigned int* solution = (unsigned int*) (char*) d_solution ;
    // // solution is of the form [a,b] where a<b and each number
    // // is an index of a queen into the 1-dimensional n*n-element chessboard
    // for (int q = 0 ; q < numQueens; q++){
    //     solution[q] = queensList[q];
    // }
  }
  