
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>

// function to add the elements of two arrays
__global__
void add(int n, float *x, float *y)
{
  //int index = 0;
  //int stride = 1;
  int index = threadIdx.x;
  int stride = blockDim.x;
  for (int i = index; i < n; i += stride)
      y[i] = x[i] + y[i];
}

int main(void)
{
  int N = 1<<20; // 1M elements

  //variable defition on system memory
  //float *x = new float[N];
  //float *y = new float[N];
 
  float *x, *y;
  //variable allocation on GPU memory
  hipMallocManaged (&x, N*sizeof(float));
  hipMallocManaged (&y, N* sizeof(float));



  
  // initialize x and y arrays on the host
  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  // Run kernel on 1M elements on the CPU
  //add(N, x, y);
  
  // Run kernel on 1M parallel elements on the GPU  
  add<<<1,256>>>(N, x, y);

  // wait for the GPU to finish the results
  hipDeviceSynchronize();

  // Check for errors (all values should be 3.0f)
  float maxError = 0.0f;
  for (int i = 0; i < N; i++)
    maxError = fmax(maxError, fabs(y[i]-3.0f));
  std::cout << "Max error: " << maxError << std::endl;

  // Free memory
  //delete [] x;
  //delete [] y;
  
  // Free GPU memory
  hipFree(x);
  hipFree(y);


  
  return 0;
}
