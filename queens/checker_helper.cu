
#include <hip/hip_runtime.h>
__device__
int countBits(unsigned int v) {
    int c; // c accumulates the total bits set in v
    for (c = 0; v; v >>= 1) {
        c += v & 1;
    }
    return c;
}

__device__
bool checkerFunc (unsigned int* queensList,int width, int numQueens){
    ////input exceptions
    // if (numQueens > width){
    //     printf("The Number of Queens is greater than width of the board\n");
    // }
    bool ifCheck = true;
    ////max we can do is 32 x 32
    unsigned int in_checkArr[32] ={0};
    for (int q = 0; q < numQueens; q++){
        int posqueen = queensList[q];
        // if (posqueen < 0 || posqueen >= width * width){
        //     printf("The position  of Queen is invalid\n");
        // }
        int row = posqueen/width;
        int col = posqueen % width;
        ////row easy!
        in_checkArr[row] |=  0xffffffff;
        for (int r = 0; r < width; r++){
            ////column in loop
            in_checkArr[r] |=  1 << col;
            ////main diagon
            if (row + col - r < width && row + col -r >= 0) 
                in_checkArr[r] |=  1 << row + col - r;
            ////other diagon
            if (col - row + r >= 0 && col - row + r < width) 
                in_checkArr[r] |=1 << col - row + r;
            if (countBits(in_checkArr[r]) < width && q == numQueens - 1)
                ifCheck = false;
        }
    }

    return ifCheck; 
}

__device__
int addtoSolution (unsigned int* queensList, int numQueens, unsigned int* solution, unsigned int* count, int pitch){
    // claim one of the valid solutions
    int solution_id = atomicAdd(count, 1);
    // printf("count = %d\n", solution_id);

    // // the below line sets solution = d_solution[solution_id]

    // solution is of the form [a,b] where a<b and each number
    // is an index of a queen into the 1-dimensional n*n-element chessboard
    
    for (int q = 0 ; q < numQueens; q++){
        solution[q] = queensList[q];
    }
    return solution_id;
    // return solution;
}