#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/sort.h>
#include <thrust/copy.h>
#include <algorithm>
#include <cstdlib>

int main(void)
{

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    // generate 32M random numbers serially
    thrust::host_vector<int> h_vec(16 << 20);
    std::generate(h_vec.begin(), h_vec.end(), rand);

    // transfer data to the device
    thrust::device_vector<int> d_vec = h_vec;

    float elapsedTime;
    hipDeviceSynchronize();
    hipEventRecord(start, 0);

    // sort data on the device (846M keys per second on GeForce GTX 480)
    thrust::sort(d_vec.begin(), d_vec.end());

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);

    // transfer data back to host
    thrust::copy(d_vec.begin(), d_vec.end(), h_vec.begin());

    hipEventDestroy(start);
    hipEventDestroy(stop);


    printf("Processing time: %f (ms)\n", elapsedTime);

    return 0;
}