#include "hip/hip_runtime.h"

/* thanks old college classmate Sean Anderson!
 * http://graphics.stanford.edu/~seander/bithacks.html#DetermineIfPowerOf2
 */

//  #include <math.h>

//  #define BOARDSIZE 4
//  #define NUMQUEENS 2
 
//  #define BOARDSIZE 12
//  #define NUMQUEENS 6

 #include "checker_helper.cu"
#include "comb_creator.cu"
 
__global__
void qgdKernelOne(int n, int a, int pitch,
               unsigned int * d_solutions, unsigned int * count, unsigned long int comb) {
    // this kernel is completely hardcoded to the 4x4 board
    // I'm not pretending otherwise

    unsigned long long int tid = threadIdx.x + blockIdx.x * blockDim.x;

  
    // we know that for n=4, a=2, so knock out all boards where a != 2
    // int bitcount = countBits(tid);
    // if (bitcount != a) { return; }

    if (tid >= comb) return;
    int width = n;

    int numQueens = a;

    // create queens List for n = 4 the size of the proposed solution is 2
    unsigned int queensList[16] = {0};

    
    queenGen (queensList, tid, width * width, numQueens);
    if (checkerFunc (queensList, width, numQueens)) 
        addSolution (queensList,  numQueens, d_solutions, count, pitch);
  
}

  __global__
  void qgdKernel(int n, int a, int pitch,
                 unsigned int * d_solutions, unsigned int * count, unsigned long int comb) {
      // this kernel is completely hardcoded to the 4x4 board
      // I'm not pretending otherwise
  
      unsigned long long int tid = threadIdx.x + blockIdx.x * blockDim.x;

    
      // we know that for n=4, a=2, so knock out all boards where a != 2
      // int bitcount = countBits(tid);
      // if (bitcount != a) { return; }
  
      if (tid >= comb) return;
      int width = n;

      int numQueens = a;

      // create queens List for n = 4 the size of the proposed solution is 2
      unsigned int queensList[16] = {0};

      
      queenGen (queensList, tid, width * width, numQueens);
      if (checkerFunc (queensList, width, numQueens)) 
          addSolution (queensList,  numQueens, d_solutions, count, pitch);
    
  }


  // Store your solutions in d_solution, which has already been allocated for you
  void qgd(int n, int a, bool one, bool all, int pitch,
           unsigned long long numSolutions, unsigned int * d_solutions,
           unsigned int * count) {
    
      // there are 2^16 possible configurations of queens on a 4x4 chessboard
      // 2^8 blocks of 2^8 threads each will check them all (brute force)
      int width = n;
      int numQueens = a;
      // TODO:
      int blockSize = 1<<10;
      //dynamic block size
      int numBlocks = 1<<6;


      //all combinations
      printf("fact ");
      unsigned long long int comb = 1;
      unsigned long long int threads;
      int m = width*width;
      int k = numQueens;
      for (int j = m ; j > m - k; j--){
            comb = comb * j;
      }
      for (int q = k; q > 0; q--){
          comb = comb / q;
      }
  
      printf("The number of the total combinations is %u\n", comb);

      threads = numBlocks * blockSize;

      printf("The number of the total threads is %u\n", threads);

      // printf("Total threads %llu\n", threads);
      //generate every possible combinations on the memory
    

      if (all == true && one == false )
        qgdKernel<<< numBlocks, blockSize >>>(width, numQueens, pitch, d_solutions, count, comb);
      else if (all == false && one == true)
        qgdKernelOne<<< numBlocks, blockSize >>>(width, numQueens, pitch, d_solutions, count, comb);

  }
  