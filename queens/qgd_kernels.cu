#include "hip/hip_runtime.h"

/* thanks old college classmate Sean Anderson!
 * http://graphics.stanford.edu/~seander/bithacks.html#DetermineIfPowerOf2
 */

 #include "checker_helper.cu"
 #include <math.h>

//  #define BOARDSIZE 6
//  #define NUMQUEENS 3
 
//  #define BOARDSIZE 4
//  #define NUMQUEENS 2
 
  __global__
  void qgdKernel(int n, int a, bool one, bool all, int pitch,
                 unsigned int * d_solutions, unsigned int * count) {
      // this kernel is completely hardcoded to the 4x4 board
      // I'm not pretending otherwise
  
      int tid = threadIdx.x + blockIdx.x * blockDim.x;

    if ( all == true  ){
        // we know that for n=4, a=2, so knock out all boards where a != 2
        int bitcount = countBits(tid);
        if (bitcount != a) { return; }
    
        int width = n;

        int numQueens = a;
        // create queens List for n = 4 the size of the proposed solution is 2
        unsigned int queensList[16] = {0};
        int temp = tid;
        for (unsigned int c = 0, qi = 0 ; temp ; temp >>= 1, c++){
            if (temp & 1){
                queensList[qi] = c;
                qi++;
            }
        }
        // unsigned int list[NUMQUEENS] = {   0	,   2	 ,  3	,  16	 , 51};

        // for (int q=0; q< numQueens; q++)
        //     queensList[q] = list[q];

        // if (checkerFunc (queensList, width, numQueens)) 
        //     // addSolution (queensList,  numQueens, d_solutions, count, pitch);
        //     printf("Solution\n");
        // else
        //     printf("This is not a Solution\n");

        if (checkerFunc (queensList, width, numQueens)) {
            addSolution (queensList,  numQueens, d_solutions, count, pitch);
            
        }
    }
  }
  
  // Store your solutions in d_solution, which has already been allocated for you
  void qgd(int n, int a, bool one, bool all, int pitch,
           unsigned long long numSolutions, unsigned int * d_solutions,
           unsigned int * count) {
    
      // there are 2^16 possible configurations of queens on a 4x4 chessboard
      // 2^8 blocks of 2^8 threads each will check them all (brute force)
      int width = n;
      int numQueens = a;
      qgdKernel<<< 1<<8, 1<<8 >>>(width, numQueens, one, all, pitch, d_solutions, count);
    //   qgdKernel<<< 1<<10, 1<<6 >>>(n, a, one, all, pitch, d_solutions, count);
    
  }
  