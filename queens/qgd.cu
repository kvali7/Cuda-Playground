#include <stdlib.h>
#include <stdio.h>
#include <string.h>


// includes, project
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "qgd_kernels2.cu"

// #define BOARDSIZE 4
// #define NUMQUEENS 2

// #define ALL 1
// #define ONE 0

#define CUDA_SAFE_CALL_NO_SYNC( call) do {                              \
    hipError_t err = call;                                                 \
    if( hipSuccess != err) {                                             \
      fprintf(stderr, "Cuda error in file '%s' in line %i : %s.\n",       \
                  __FILE__, __LINE__, hipGetErrorString( err) );         \
      exit(EXIT_FAILURE);                                                 \
      } } while (0)
  
  #define CUDA_SAFE_CALL( call) do {                                      \
    CUDA_SAFE_CALL_NO_SYNC(call);                                         \
    hipError_t err = hipDeviceSynchronize();                              \
    if( hipSuccess != err) {                                             \
       fprintf(stderr, "Cuda error in file '%s' in line %i : %s.\n",      \
                   __FILE__, __LINE__, hipGetErrorString( err) );        \
       exit(EXIT_FAILURE);                                                \
       } } while (0)
  
  void usage(char** argv) {
      fprintf(stderr, "Usage: %s -{one,all} n a\n", argv[0]);
      exit(1);
  }
  
  unsigned long long solutionsTable[] = {
    0, 1, 4, 1, 12, 186, 4, 86, 4860, 114, 8, 2, 8, 288, 500, 500 , 500, 500 , 500, 500, 500, 500
    // https://oeis.org/A002564   (this is a(n))
    // we have no numbers past this, please post them as you find them
    // you will need to fill in this array as n gets bigger
};
  
  int main(int argc, char** argv) {
  
  
      //args
        if (argc != 4) {
            usage(argv);
        }
        bool one = false;
        bool all = false;
        if (!strcmp(argv[1], "-one")) {
            one = true;
        }
        if (!strcmp(argv[1], "-all")) {
            all = true;
        }
        if (!one && !all) {
            usage(argv);
        }
        int n = atoi(argv[2]);
        int a = atoi(argv[3]);
  
      unsigned long long numSolutions = solutionsTable[n];
  
      hipEvent_t start, stop;
      hipEventCreate(&start);
      hipEventCreate(&stop);
  
      // allocate memory
      unsigned int* h_solutions;
      h_solutions = (unsigned int*)malloc(numSolutions * a * sizeof(unsigned int));
      unsigned int* d_solutions;
      unsigned int* count;
      hipMallocManaged(&count, sizeof(unsigned int));
      *count = 0;
      size_t dpitch;
      // height is number of solutions
      // width is items per solution
      CUDA_SAFE_CALL(hipMallocPitch(&d_solutions, &dpitch,
          a * sizeof(unsigned int),
          numSolutions));
      float elapsedTime;
      hipDeviceSynchronize();
      hipEventRecord(start, 0);
      qgd(n, a, one, all, dpitch, numSolutions, d_solutions, count);
      hipEventRecord(stop, 0);
      hipEventSynchronize(stop);
      hipEventElapsedTime(&elapsedTime, start, stop);
  
      size_t hpitch = a * sizeof(unsigned int);
  
      CUDA_SAFE_CALL(hipMemcpy2D(h_solutions, hpitch, d_solutions, dpitch,
          a * sizeof(unsigned int), // width
          numSolutions,                // height
          hipMemcpyDeviceToHost));
  
      hipEventDestroy(start);
      hipEventDestroy(stop);

    //   unsigned long long int valcount;


    //   // print answers
      int maxi = 0;
      if (all == true && one ==false)
        maxi = numSolutions;
      else
        maxi = 1;

        for (int i = 0; i < maxi; i++) {
            unsigned int* solution = &h_solutions[i * a];
            // i is which solution
            printf("Solution %d is [", i);
            for (int k = 0; k < a; k++) {
                printf("%d%s", solution[k], (k < a - 1) ? ", " : "]");
            }
            for (int j = 0, k = 0; j < n * n; j++) {
                // j is which square of the chessboard (note each solution
                // must be in sorted order)
                // k is which queen we're placing (also index into the soln. array)
                if (j % n == 0) {
                    printf("\n");
                }
                if (j == solution[k] && k < a) {
                    printf("Q");
                    k++;
                }
                else {
                    printf(".");
                }
            }
            printf("\n");
        }


        if (*count == numSolutions)
        printf("All %llu solutions found!\n", numSolutions);
    else
        printf("Only %u solutions\n", *count);
  
        printf("Processing time: %f (ms)\n", elapsedTime);

  
      // clean up memory
      free(h_solutions);
      CUDA_SAFE_CALL(hipFree(d_solutions));
      hipFree(count);
  }
  