
#include <hip/hip_runtime.h>


__device__
int countBits(unsigned int v) {
    int c; // c accumulates the total bits set in v
    for (c = 0; v; v >>= 1) {
        c += v & 1;
    }
    return c;
}

__device__
bool checkerFunc (int* queensList,int width, int numQueens){
    ////input exceptions
    // if (numQueens > width){
    //     printf("The Number of Queens is greater than width of the board\n");
    // }
    bool ifCheck = true;
    ////max we can do is 32 x 32
    unsigned int in_checkArr[32] ={0};
    for (int queen = 0; queen < numQueens; queen++){
        int posqueen = queensList[queen];
        // if (posqueen < 0 || posqueen >= width * width){
        //     printf("The position  of Queen is invalid\n");
        // }
        int row = posqueen/width;
        int col = posqueen % width;
        ////row easy!
        in_checkArr[row] |=  0xffffffff;
        for (int r = 0; r < width; r++){
            ////column in loop
            in_checkArr[r] |=  1 << col;
            ////main diagon
            if (row + col - r < width && row + col -r >= 0) 
                in_checkArr[r] |=  1 << row + col - r;
            ////other diagon
            if (col - row + r >= 0 && col - row + r < width) 
                in_checkArr[r] |=1 << col - row + r;
            if (countBits(in_checkArr[r]) < width && queen == numQueens - 1)
                ifCheck = false;
        }
    }

    return ifCheck; 
}
