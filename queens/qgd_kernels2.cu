#include "hip/hip_runtime.h"

/* thanks old college classmate Sean Anderson!
 * http://graphics.stanford.edu/~seander/bithacks.html#DetermineIfPowerOf2
 */


 #include "checker_helper.cu"
#include "comb_creator.cu"
 
__global__
void qgdKernelOne(int n, int a, int pitch,
    unsigned int* d_solution, unsigned int* count, unsigned long int comb, unsigned int* flag) {

    unsigned long long int tid = threadIdx.x + blockIdx.x * blockDim.x;
    // int stride = blockDim.x * gridDim.x;
    // for (unsigned long long int id = tid; id < 10 * tid; id += stride)

        // if (tid >= comb || *flag) return;
        if (*flag) return;
        int width = n;

        int numQueens = a;

        // create queens List of max size 16
        unsigned int queensList[16] = {0};
        queenGen (queensList, tid, width * width, numQueens);
        if (!checkerFunc (queensList, width, numQueens) ) return;
          addtoSolution (queensList,  numQueens, d_solution, count, pitch);
          int valflag = atomicAdd(flag, 1);
          // printf("flag = %d\n",valflag);
  
}

  __global__
  void qgdKernel(int n, int a, int pitch,
        unsigned int* d_solution, unsigned int* count, unsigned long int comb) {

  
      unsigned long long int tid = threadIdx.x + blockIdx.x * blockDim.x;
      // int stride = blockDim.x * gridDim.x;
      // for (unsigned long long int id = tid; id < 10 * tid; id += stride){
        
          // if (tid >= comb) return;
          int width = n;

          int numQueens = a;

          // create queens List of max size 16
          unsigned int queensList[16] = {0};
          queenGen (queensList, tid, width * width, numQueens);
          if (!checkerFunc (queensList, width, numQueens)) return;
            addtoSolution (queensList,  numQueens, d_solution, count, pitch);

      // }
      
  }


  // Store your solutions in d_solution, which has already been allocated for you
  void qgd(int n, int a, bool one, bool all, int pitch,
           unsigned long long numSolutions, unsigned int * d_solution,
           unsigned int * count) {
    
      // there are 2^16 possible configurations of queens on a 4x4 chessboard
      // 2^8 blocks of 2^8 threads each will check them all (brute force)
      int width = n;
      int numQueens = a;
     
      unsigned int *flag;
      hipMallocManaged(&flag, sizeof(unsigned int));
      *flag = 0;

      //all combinations
      // printf("fact ");
      unsigned long long int comb = 1;
      int m = width*width;
      int k = numQueens;
      for (int j = m ; j > m - k; j--){
            comb = comb * j;
      }
      for (int q = k; q > 0; q--){
          comb = comb / q;
      }


      unsigned long int blockSize;

      if (width < 5)
        blockSize = 1 << 5;
      else
        blockSize = 1<<10;
            //dynamic block size for all
      unsigned long long int numBlocks;

      unsigned long long int numBlocksAllTable[] = {
        1<<2, 1<<2, 1<<2, 1<<2, 1<<2, 1<<2, 1<<6, 1<<8, 1<<13, 1<<15, 1<<16,  1<<17, 1<<24, 1<<27};

      unsigned long long int numBlocksOneTable[] = {
        1<<1, 1<<1, 1<<1, 1<<1, 1<<1, 1<<1, 1<<1, 1<<4, 1<<5, 1<<10, 1<<14, 1<<17,  1<<19, 1<<24, 1<<27};
      if (all == true && one == false)
        numBlocks = comb / blockSize + 1;
      else if (one == true && all ==false)
        numBlocks = numBlocksOneTable[width];
      else
        printf("all one error!\n");

      // if (numBlocks < 1)
      //   numBlocks = 1;
      // if (all == true){

      //      n = 2   0.32
      //  n = 3   0.32
      //  n = 4   0.37
      //  n = 5   0.5 ms
      //   n = 6   0.6 ms
      //     n = 7   4 ms
      // n = 8   232 ms
      // n = 9   756 ms
      //n = 10   1 secs
      //n = 11    1.2 secs
      // n = 12   85 secs 
      // n = 13  273 secs

      // if (one == true){
      // n = 2   0.32
      //  n = 3   0.32
      //   n = 4   0.33
      //  n = 5   0.4 ms
      // n = 6   0.6 ms
      //  n = 7   0.9 ms
      //   n = 8   1 ms
      //   n = 9   11 ms
      //  n = 10   365 ms
      //   n = 11    856 ms
      //     n = 12   1.9 secs 
      //   n = 13     50 secs
      //  n = 14     325 secs
    
  
        // numBlocks = 224;

      //  blockSize = 1<<10;
      //  numBlocks = 1<<2;
  
      // printf("The number of the total combinations is %llu\n", comb);
      //  unsigned long long int threads;

      // threads = numBlocks * blockSize;

      // printf("thread = %d, blocks = %d The number of the total threads is %llu\n", blockSize, numBlocks, threads);

      // printf("Total threads %llu\n", threads);
    

      if (all == true && one == false )
        qgdKernel<<< numBlocks, blockSize >>>(width, numQueens, pitch, d_solution, count, comb);
      else if (all == false && one == true)
        qgdKernelOne<<< numBlocks, blockSize >>>(width, numQueens, pitch, d_solution, count, comb, flag);

  }
  