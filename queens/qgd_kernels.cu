
#include <hip/hip_runtime.h>

/* thanks old college classmate Sean Anderson!
 * http://graphics.stanford.edu/~seander/bithacks.html#DetermineIfPowerOf2
 */

 #define BOARDSIZE 4
 #define NUMQUEENS 2
 
 __device__
int countBits(unsigned int v, int boardSize) {
    int c; // c accumulates the total bits set in v
    for (c = 0; v; v >>= 1) {
        c += v & 1;
    }
    return c;
}
 
 
 __device__
 bool checkerFunc (int* queensList,int boardSize, int numQueens){
    // int queensList[NUMQUEENS] = {2,17,22};
     bool ifCheck = true;
     //max we can do is 32 x 32
     unsigned int in_checkArr[32] ={0};
     for (int queen = 0; queen < numQueens; queen++){
         int posqueen = queensList[queen];
         int row = posqueen/boardSize;
         int col = posqueen % boardSize;
         //row easy!
         in_checkArr[row] |=  0xffffffff;
         for (int r = 0; r < boardSize; r++){
             // column in loop
             in_checkArr[r] |=  1 << col;
             // main diagon
             if (row + col - r < boardSize && row + col -r >= 0) 
                 in_checkArr[r] |=  1 << row + col - r;
             // other diagon
             if (col - row + r >= 0 && col - row + r < boardSize) 
                 in_checkArr[r] |=1 << col - row + r;
             if (countBits(in_checkArr[r], boardSize) < boardSize && queen == numQueens - 1)
                 ifCheck = false;
         }
     }
 
     return ifCheck; 
 }
 
  __global__
  void qgdKernel(int n, int a, bool one, bool all, int pitch,
                 unsigned int * d_solution, unsigned int * count) {
      // this kernel is completely hardcoded to the 4x4 board
      // I'm not pretending otherwise
  
      int tid = threadIdx.x + blockIdx.x * blockDim.x;
  
      // tid is a 16-bit number, where each bit corresponds to 1 square
      // on the chessboard.
      // a 1
  
      // strategy: look at every single possible board in parallel
      // this is very inefficient and scales extremely poorly
      // it is a bad method
      // do not do this to scale to more chessboards
  
      // we know that for n=4, a=2, so knock out all boards where a != 2
      int bitcount = countBits(tid,17);
      if (bitcount != a) { return; }
  

      // create queens List for n = 4 the size of the proposed solution is 2
     int queensList[BOARDSIZE] = {0};
     int temp = tid;
     for (int c = 0, qi = 0 ; temp ; temp >>= 1, c++){
         if (temp & 1){
             queensList[qi] = c;
             qi++;
         }
     }
     if (!checkerFunc (queensList, BOARDSIZE, NUMQUEENS)) {return;}
 
 
    //   int in_check = 0;           // start with no squares in check
    //   int loc;
    //   for (loc = 0; loc < n*n; loc++) { // iterate over possible queen locations
    //       int queen = 1 << loc;
    //       if (tid & queen) {      // there's a queen at position i
    //           // set the whole row in check
    //           int row = 0xf << (loc & 0xc);
    //           in_check |= row;
    //           // set the whole column in check
    //           int col = 0x1111 << (loc & 0x3);
    //           in_check |= col;
    //           // if we're on a diagonal, set entire diagonal
    //           if (queen & 0x2100) { in_check |= 0x2100; }
    //           if (queen & 0x4210) { in_check |= 0x4210; }
    //           if (queen & 0x8421) { in_check |= 0x8421; }
    //           if (queen & 0x0842) { in_check |= 0x0842; }
    //           if (queen & 0x0084) { in_check |= 0x0084; }
  
    //           if (queen & 0x4800) { in_check |= 0x4800; }
    //           if (queen & 0x2480) { in_check |= 0x2480; }
    //           if (queen & 0x1248) { in_check |= 0x1248; }
    //           if (queen & 0x0124) { in_check |= 0x0124; }
    //           if (queen & 0x0012) { in_check |= 0x0012; }
    //       }
    //   }
    //   if (in_check != 0xffff) { return; }
  
      // if we've reached this point, we have a valid board with configuration tid
      // printf("%x\n", tid);
  
      // claim one of the valid solutions
      int solution_id = atomicAdd(count, 1);
  
      // the below line sets solution = d_solution[solution_id]
      unsigned int * solution =
          (unsigned int *) ((char *) d_solution + solution_id * pitch);
  
      // solution is of the form [a,b] where a<b and each number
      // is an index of a queen into the 1-dimensional n*n-element chessboard
      int i, c, k;
      for (i = 0, c = tid, k = 0; c != 0; i++, c >>= 1) {
          if (c & 1) {
              solution[k++] = i;
          }
      }
  }
  
  // Store your solutions in d_solution, which has already been allocated for you
  void qgd(int n, int a, bool one, bool all, int pitch,
           unsigned long long numSolutions, unsigned int * d_solution,
           unsigned int * count) {
      if (one) {
          fprintf(stderr, "Instructor's solution only works for -all\n");
          exit(42);
      }
      if (n != 4) {
          fprintf(stderr, "Instructor's solution only works for n=4\n");
          exit(4);
      }
      if (a != 2) {
          fprintf(stderr, "Instructor's solution only works for a=2\n");
          exit(2);
      }
  
      // there are 2^16 possible configurations of queens on a 4x4 chessboard
      // 2^8 blocks of 2^8 threads each will check them all (brute force)
      qgdKernel<<< 1<<8, 1<<8 >>>(n, a, one, all, pitch, d_solution, count);
  }
  