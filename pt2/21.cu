
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
//#include <cuda_runtime.h>


// function to copy the elements of an array and decrement to make the compiler not override it
__global__
void copyKernel(int n, float4* x, float4* y, float4* z, float4* w){

  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int i = index; i < n; i += stride){
      y[i].x = x[i].x - 1.0;
      y[i].y = x[i].y - 1.0;
      y[i].z = x[i].z - 1.0;
      y[i].w = x[i].w - 1.0;
      z[i].x = y[i].x - 1.0;
      z[i].y = y[i].y - 1.0;
      z[i].z = y[i].z - 1.0;
      z[i].w = y[i].w - 1.0;
      w[i].x = z[i].x - 1.0;
      w[i].y = z[i].y - 1.0;
      w[i].z = z[i].z - 1.0;
      w[i].w = z[i].w - 1.0;
  }
}

__global__
void init(int n, float4* x, float val){

  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int i = index; i < n; i += stride){
     x[i].x = val;
     x[i].y = val;
     x[i].z = val;
     x[i].w = val;
  }
}



int main(int argc,char* argv[]){

  int N = 1<<20;

  int blockSize = 256;
  int numBlocks = (N + blockSize - 1) / blockSize;

  if (argc >= 2)
     blockSize = atoi(argv[1]);

  if (argc >= 3)
     numBlocks = atoi(argv[2]);
  
  std::cout<<"blockSize= "<<blockSize<<std::endl;
  std::cout<<"numBlocks= "<<numBlocks<<std::endl;
 
  float4 *x, *y, *z, *w;
  //variable allocation on GPU memory
  hipMallocManaged (&x, N*sizeof(float4));
  hipMallocManaged (&y, N* sizeof(float4));
  hipMallocManaged (&z, N*sizeof(float4));
  hipMallocManaged (&w, N* sizeof(float4));
  
  // initialize x and y arrays on the device
  float val = 3.0f;
  init<<<numBlocks, blockSize>>>(N, x, val);
  
  // Run kernel on 1M parallel elements on the GPU  
  copyKernel<<<numBlocks, blockSize>>>(N, x, y, z, w);

  // wait for the GPU to finish the results
  hipDeviceSynchronize();

  // Check for errors (all values should be 3.0f)
  float maxError = 0.0f;
  for (int i = 0; i < N; i++){
    maxError = fmax(maxError, fabs(y[i].x - 2.0f));
    maxError = fmax(maxError, fabs(y[i].y - 2.0f));
    maxError = fmax(maxError, fabs(y[i].z - 2.0f));
    maxError = fmax(maxError, fabs(y[i].w - 2.0f));
    maxError = fmax(maxError, fabs(z[i].x - 1.0f));
    maxError = fmax(maxError, fabs(z[i].y - 1.0f));
    maxError = fmax(maxError, fabs(z[i].z - 1.0f));
    maxError = fmax(maxError, fabs(z[i].w - 1.0f));
    maxError = fmax(maxError, fabs(w[i].x - 0.0f));
    maxError = fmax(maxError, fabs(w[i].y - 0.0f));
    maxError = fmax(maxError, fabs(w[i].z - 0.0f));
    maxError = fmax(maxError, fabs(w[i].w - 0.0f));
}
  std::cout << "Max error: " << maxError << std::endl;

  // Free GPU memory
  hipFree(x);
  hipFree(y);
  hipFree(z);
  hipFree(w);


  return 0;
}
