
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>

typedef struct 
{
  double x;
  double y;
  double z;
  double w;
  double xx;
  double yy;
  double zz;
  double ww;
} double8;

// Kernel function to add the elements of two arrays
__global__
void cpy_double8(int n, double8 *src, double8 *dst)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int i = index; i < n; i += stride){
    dst[i].x = src[i].x + 1;
    dst[i].y = src[i].y + 1;
    dst[i].z = src[i].z + 1;
    dst[i].w = src[i].w + 1;
    dst[i].xx = src[i].xx + 1;
    dst[i].yy = src[i].yy + 1;
    dst[i].zz = src[i].zz + 1;
    dst[i].ww = src[i].ww + 1;
  }
}

__global__
void fill_double8(int n, double8 *dst)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int i = index; i < n; i += stride){
    dst[i].x = 0;
    dst[i].y = 0;
    dst[i].z = 0;
    dst[i].w = 0;
    dst[i].xx = 0;
    dst[i].yy = 0;
    dst[i].zz = 0;
    dst[i].ww = 0;
  }
}

int main(void)
{
  int N = 1 << 20;
  double8 *x, *y, *z;
  
  // Allocate Unified Memory – accessible from CPU or GPU
  hipMallocManaged(&x, N * sizeof(double8));
  hipMallocManaged(&y, N * sizeof(double8));
  hipMallocManaged(&z, N * sizeof(double8));

  int blockSize = 256;
  int numBlocks = (N + blockSize - 1) / blockSize;

  /*
  // initialize x and y arrays on the host
  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }
  */
  //fill_double8<<<numBlocks, blockSize>>>(N, x);
  
  /*
  // Run kernel on 1M elements on the GPU
  int blockSize = 256;
  int numBlocks = (N + blockSize - 1) / blockSize;
  add<<<numBlocks, blockSize>>>(N, x, y);
  */
  cpy_double8<<<numBlocks, blockSize>>>(N, x, y);

  // Wait for GPU to finish before accessing on host
  hipDeviceSynchronize();

  // Copying to host memory 
  hipMemcpy(z, y, N * sizeof(double8), hipMemcpyDeviceToHost);

  double maxError = 0.0;
  for (int i = 0; i < N; i++){
    maxError = fmax(maxError, fabs(z[i].x - 1.0));
    maxError = fmax(maxError, fabs(z[i].y - 1.0));
    maxError = fmax(maxError, fabs(z[i].z - 1.0));
    maxError = fmax(maxError, fabs(z[i].w - 1.0));
    maxError = fmax(maxError, fabs(z[i].xx - 1.0));
    maxError = fmax(maxError, fabs(z[i].yy - 1.0));
    maxError = fmax(maxError, fabs(z[i].zz - 1.0));
    maxError = fmax(maxError, fabs(z[i].ww - 1.0));
  }
  std::cout << "Max error: " << maxError << std::endl;

  // Free memory
  hipFree(x);
  hipFree(y);
  free(z);
  return 0;
}
