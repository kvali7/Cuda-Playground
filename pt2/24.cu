
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
//#include <cuda_runtime.h>


// function to copy the elements of an array and decrement to the compiler not override it
__global__
void newtonKernel(int n, float4* x, float4* y, float4* z){
	float4 result = make_float4 (1.0f,1.0f,1.0f,1.0f);
        int index = blockIdx.x * blockDim.x + threadIdx.x;
        int stride = blockDim.x * gridDim.x;
        for (int i = index; i < n; i += stride){

		for (int k =0; k < 40; k++){
			result.x = result.x * k * x[i].x * y[i].x + x[i].x + y[i].x;
			result.y = result.y * k * x[i].y * y[i].y + x[i].y + y[i].y;
			result.z = result.z * k * x[i].z * y[i].z + x[i].z + y[i].z;
			result.w = result.w * k * x[i].w * y[i].w + x[i].w + y[i].w;
		}
	      z[i] = result  ;
	      y[i].x = z[i].x ;
	      y[i].y = z[i].y ;
	      y[i].z = z[i].z ;
	      y[i].w = z[i].w ;
	      x[i].x = y[i].x ;
	      x[i].y = y[i].y ;
	      x[i].z = y[i].z ;
	      x[i].w = y[i].w ;

  	}
}


int main(void){

  int N = 1<<20;

  int blockSize = 256;
  int numBlocks = (N + blockSize - 1) / blockSize;
 
  float4 *x, *y, *z;
  //variable allocation on GPU memory
  hipMallocManaged (&x, N*sizeof(float4));
  hipMallocManaged (&y, N* sizeof(float4));
  hipMallocManaged (&z, N*sizeof(float4));

  
  // initialize x and y arrays on the device
  //float val = 3.0f;

  // initialize x and y arrays on the host
  for (int i = 0; i < N; i++) {
    x[i] = make_float4(1.0f,1.0f,1.0f, 1.0f);
    y[i] = make_float4(2.0f,2.0f,2.0f, 2.0f);
    z[i] = make_float4(1.0f,1.0f,1.0f, 1.0f);;
  }

  // Run kernel on 1M parallel elements on the GPU  
  newtonKernel<<<numBlocks, blockSize>>>(N, x, y, z);

  // wait for the GPU to finish the results
  hipDeviceSynchronize();

  // Check for errors (all values should be 3.0f)
  float maxError = 0.0f;
  for (int i = 0; i < N; i++){
   maxError = fmax(maxError, fabs(x[i].x - 1.0f));
    maxError = fmax(maxError, fabs(x[i].y - 1.0f));
    maxError = fmax(maxError, fabs(x[i].z - 1.0f));
    maxError = fmax(maxError, fabs(x[i].w - 1.0f));
   maxError = fmax(maxError, fabs(y[i].x - 1.0f));
    maxError = fmax(maxError, fabs(y[i].y - 1.0f));
    maxError = fmax(maxError, fabs(y[i].z - 1.0f));
    maxError = fmax(maxError, fabs(y[i].w - 1.0f));
    maxError = fmax(maxError, fabs(z[i].x - 1.0f));
    maxError = fmax(maxError, fabs(z[i].y - 1.0f));
    maxError = fmax(maxError, fabs(z[i].z - 1.0f));
    maxError = fmax(maxError, fabs(z[i].w - 1.0f));
}
  std::cout << "Max error: " << maxError << std::endl;

  // Free GPU memory
  hipFree(x);
  hipFree(y);
  hipFree(z);



  return 0;
}
