
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
//#include <cuda_runtime.h>


// function to copy the elements of an array and decrement to the compiler not override it
__global__
void newtonKernel(int n, float4* x, float4* y, float4* z){
	float4 result = make_float4 (1.0f,1.0f,1.0f,1.0f);
        int index = blockIdx.x * blockDim.x + threadIdx.x;
        int stride = blockDim.x * gridDim.x;
        for (int i = index; i < n; i += stride){

		for (int k = 1;k<600;k++){

		result.x = result.x*(k*x[i].x +k*y[i].x + k*result.x + k*x[i].x * k*y[i].x + k*x[i].x * k*result.x + k*x[i].x * k*x[i].x + k*y[i].x * k*result.x + k*x[i].x * k*y[i].x * k*result.x) ;
   		result.y = result.y*(k*x[i].y +k*y[i].y + k*result.y + k*x[i].y * k*y[i].y + k*x[i].y * k*result.y + k*x[i].y * k*x[i].y + k*y[i].y * k*result.y + k*x[i].y * k*y[i].y * k*result.y) ;
      		result.z = result.z*(k*x[i].z +k*y[i].z + k*result.z + k*x[i].z * k*y[i].z + k*x[i].z * k*result.z + k*x[i].z * k*x[i].z + k*y[i].z * k*result.z + k*x[i].z * k*y[i].z * k*result.z)  ;
      		result.w = result.w*(k*x[i].w +k*y[i].w + k*result.w + k*x[i].w * k*y[i].w + k*x[i].w * k*result.w + k*x[i].w * k*x[i].w + k*y[i].w * k*result.w + k*x[i].w * k*y[i].w * k*result.w)  ;

	  	}
		z[i] = result ;

  	}
}


int main(void){

  int N = 1<<20;

  int blockSize = 1024;
  int numBlocks = (N + blockSize - 1) / blockSize;
 
  float4 *x, *y, *z;
  //variable allocation on GPU memory
  hipMallocManaged (&x, N*sizeof(float4));
  hipMallocManaged (&y, N* sizeof(float4));
  hipMallocManaged (&z, N*sizeof(float4));

  
  // initialize x and y arrays on the device
  //float val = 3.0f;

  // initialize x and y arrays on the host
  for (int i = 0; i < N; i++) {
    x[i] = make_float4(1.0f,1.0f,1.0f, 1.0f);
    y[i] = make_float4(2.0f,2.0f,2.0f, 2.0f);
    z[i] = make_float4(1.0f,1.0f,1.0f, 1.0f);;
  }

  // Run kernel on 1M parallel elements on the GPU  
  newtonKernel<<<numBlocks, blockSize>>>(N, x, y, z);

  // wait for the GPU to finish the results
  hipDeviceSynchronize();

  // Check for errors (all values should be 3.0f)
  float maxError = 0.0f;
  for (int i = 0; i < N; i++){

    maxError = fmax(maxError, fabs(z[i].x - 1.0f));
    maxError = fmax(maxError, fabs(z[i].y - 1.0f));
    maxError = fmax(maxError, fabs(z[i].z - 1.0f));
    maxError = fmax(maxError, fabs(z[i].w - 1.0f));
}
  std::cout << "Max error: " << maxError << std::endl;

  // Free GPU memory
  hipFree(x);
  hipFree(y);
  hipFree(z);



  return 0;
}
