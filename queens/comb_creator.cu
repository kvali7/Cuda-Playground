
#include <hip/hip_runtime.h>
__device__ //mCk
int combin(int m, int k) {
    unsigned long int comb = 1;
    if (m > 0 && k > 0) {
        for (int j = m; j > m - k; j--) {
            comb = comb * j;
        }
        for (int q = k; q > 0; q--) {
            comb = comb / q;
        }
    }
    return comb;
}

__device__
void queenGen(unsigned int* queensList, unsigned long long int i, int m, int k) {
    int q = 0;
    unsigned long long int r = i + 1;
    int j = 0;
    for (int s = 1; s < k + 1; s++) {
        int cs = j + 1;
        int com = combin(m - cs, k - s);
        while (r > com) {
            r -= com;
            cs += 1;
            com = combin(m - cs, k - s);
        }
        queensList[q++] = cs - 1;
        j = cs;
    }
}

