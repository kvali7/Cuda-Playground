#include "hip/hip_runtime.h"

/* thanks old college classmate Sean Anderson!
 * http://graphics.stanford.edu/~seander/bithacks.html#DetermineIfPowerOf2
 */


 #include "checker_helper.cu"
#include "comb_creator.cu"
 
__global__
void qgdKernelOne(int n, int a, int pitch,
    unsigned int * d_solution, unsigned int * count, unsigned long int comb, unsigned int* flag) {

    unsigned long long int tid = threadIdx.x + blockIdx.x * blockDim.x;
    // int stride = blockDim.x * gridDim.x;
    // for (unsigned long long int id = tid; id < 10 * tid; id += stride){
  
        // we know that for n=4, a=2, so knock out all boards where a != 2
        // int bitcount = countBits(tid);
        // if (bitcount != a) { return; }
        if (tid >= comb || *flag) return;
        int width = n;

        int numQueens = a;

        // create queens List of max size 16
        unsigned int queensList[16] = {0};
        queenGen (queensList, tid, width * width, numQueens);
        if (!checkerFunc (queensList, width, numQueens) ) return;
          addtoSolution (queensList,  numQueens, d_solution, count, pitch);
          int valflag = atomicAdd(flag, 1);
          // printf("flag = %d\n",valflag);


    
        
  
}

  __global__
  void qgdKernel(int n, int a, int pitch,
        unsigned int * d_solution, unsigned int * count, unsigned long int comb) {

  
      unsigned long long int tid = threadIdx.x + blockIdx.x * blockDim.x;
      // int stride = blockDim.x * gridDim.x;
      // for (unsigned long long int id = tid; id < 10 * tid; id += stride){
    
          // we know that for n=4, a=2, so knock out all boards where a != 2
          // int bitcount = countBits(tid);
          // if (bitcount != a) { return; }
      
          if (tid >= comb) return;
          int width = n;

          int numQueens = a;

          // create queens List of max size 16
          unsigned int queensList[16] = {0};
          queenGen (queensList, tid, width * width, numQueens);
          if (!checkerFunc (queensList, width, numQueens)) return;
            addtoSolution (queensList,  numQueens, d_solution, count, pitch);
      
  }


  // Store your solutions in d_solution, which has already been allocated for you
  void qgd(int n, int a, bool one, bool all, int pitch,
           unsigned long long numSolutions, unsigned int * d_solution,
           unsigned int * count) {
    
      // there are 2^16 possible configurations of queens on a 4x4 chessboard
      // 2^8 blocks of 2^8 threads each will check them all (brute force)
      int width = n;
      int numQueens = a;
      // TODO:
      unsigned long int blockSize = 1<<10;
            //dynamic block size for all
        unsigned long int numBlocks;
      if (all == true){
        switch (width){
          case 2:
           numBlocks= 1<<6; 

            //  n = 2   0.25 ms
          case 3:
           numBlocks= 1<<6; 


      //  n = 3   0.25 ms
        case 4:
         numBlocks= 1<<6; 

        //  n = 4   0.25 ms
        case 5:
         numBlocks= 1<<6; 

        // n = 5   0.5 ms
        case 6:

          numBlocks= 1<<6; //until n = 6   0.6 ms
        case 7:

         numBlocks= 1<<8; //until n = 7   4 ms
        case 8:

          numBlocks= 1<<13; //until n = 8   232 ms
        case 9:

          numBlocks= 1<<15; //until n = 9   756 ms
        case 10:

          numBlocks= 1<<16; //until n = 10   1 secs
        case 11:

          numBlocks= 1<<17; //until n = 11    1.2 secs
        case 12:

          numBlocks= 1<<24; //until n = 12   85 secs 
        case 13:

         numBlocks= 1<<27; //until n = 13 273 secs
        // case default:
        //  numBlocks= 1<<29; //until n = 13 273 secs

      }
      }
            //dynamic block size for one

      if (one == true){
        switch (width){
          case 2:
           numBlocks= 1<<1;
            // n = 2   0.25 ms
            case 3:
             numBlocks= 1<<1;
          //  n = 3   0.25 ms
            case 4:
             numBlocks= 1<<1;
          //  n = 4   0.25 ms
            case 5:
             numBlocks= 1<<1;
            // n = 5   0.4 ms
            case 6:

             numBlocks= 1<<1; //until n = 6   0.6 ms
            case 7:

             numBlocks= 1<<4; //until n = 7   0.9 ms
            case 8:

            numBlocks= 1<<5; //until n = 8   1 ms
            case 9:

             numBlocks= 1<<10; //until n = 9   22 ms
            case 10:

            numBlocks= 1<<14; //until n = 10   365 ms
            case 11:

             numBlocks= 1<<17; //until n = 11    856 ms
            case 12:

             numBlocks= 1<<19; //until n = 12   1.9 secs 
            case 13:

         numBlocks= 1<<24; //until n = 13     50 secs
        case 14:

         numBlocks= 1<<29; //until n = 14     325 secs

          // case default:
          //  numBlocks= 1<<30; //until n = 13  

        }
      }

  
      unsigned int *flag;
      hipMallocManaged(&flag, sizeof(unsigned int));
      *flag = 0;

      //all combinations
      // printf("fact ");
      unsigned long long int comb = 1;
      // unsigned long long int threads;
      int m = width*width;
      int k = numQueens;
      for (int j = m ; j > m - k; j--){
            comb = comb * j;
      }
      for (int q = k; q > 0; q--){
          comb = comb / q;
      }
  
      // printf("The number of the total combinations is %llu\n", comb);

      // threads = numBlocks * blockSize;

      // printf("The number of the total threads is %llu\n", threads);

      // printf("Total threads %llu\n", threads);
      //generate every possible combinations on the memory
    

      if (all == true && one == false )
        qgdKernel<<< numBlocks, blockSize >>>(width, numQueens, pitch, d_solution, count, comb);
      else if (all == false && one == true)
        qgdKernelOne<<< numBlocks, blockSize >>>(width, numQueens, pitch, d_solution, count, comb, flag);

  }
  