// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <iostream>

// includes, project
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// #include "qgd_kernels.cu"
#include "qgd_kernels2.cu"

#define CUDA_SAFE_CALL_NO_SYNC( call) do {                              \
  hipError_t err = call;                                                 \
  if( hipSuccess != err) {                                             \
    fprintf(stderr, "Cuda error in file '%s' in line %i : %s.\n",       \
                __FILE__, __LINE__, hipGetErrorString( err) );         \
    exit(EXIT_FAILURE);                                                 \
    } } while (0)

#define CUDA_SAFE_CALL( call) do {                                      \
  CUDA_SAFE_CALL_NO_SYNC(call);                                         \
  hipError_t err = hipDeviceSynchronize();                              \
  if( hipSuccess != err) {                                             \
     fprintf(stderr, "Cuda error in file '%s' in line %i : %s.\n",      \
                 __FILE__, __LINE__, hipGetErrorString( err) );        \
     exit(EXIT_FAILURE);                                                \
     } } while (0)

void usage(char ** argv) {
    fprintf(stderr, "Usage: %s -{one,all} n a\n", argv[0]);
    exit(1);
}

unsigned long long solutionsTable[] = {
    0, 1, 4, 1, 12, 186, 4, 86, 4860, 114, 8, 2, 8, 288
    // https://oeis.org/A002564   (this is a(n))
    // we have no numbers past this, please post them as you find them
    // you will need to fill in this array as n gets bigger
};

int main(int argc, char** argv) {
    if (argc != 4) {
        usage(argv);
    }
    bool one = false;
    bool all = false;
    if (!strcmp(argv[1], "-one")) {
        one = true;
    }
    if (!strcmp(argv[1], "-all")) {
        all = true;
    }
    if (!one && !all) {
        usage(argv);
    }
    int n = atoi(argv[2]);
    int a = atoi(argv[3]);

    // n = 4;
    // a = 2;

    // all = true;
    // one = false;

    unsigned long long numSolutions = solutionsTable[n];

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // allocate memory
    unsigned int * h_solutions;
    h_solutions = (unsigned int *) malloc(numSolutions * a * sizeof(unsigned int));
    unsigned int * d_solutions;
    unsigned int * count;
    hipMallocManaged(&count, sizeof(unsigned int));
    *count = 0;
    size_t dpitch;
    // height is number of solutions
    // width is items per solution
    CUDA_SAFE_CALL(hipMallocPitch(&d_solutions, &dpitch,
                                   a * sizeof(unsigned int),
                                   numSolutions));
    float elapsedTime;
    hipDeviceSynchronize();
    hipEventRecord(start, 0);
    qgd(n, a, one, all, dpitch, numSolutions, d_solutions, count);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);

    size_t hpitch = n * sizeof(unsigned int);
    CUDA_SAFE_CALL(hipMemcpy2D(h_solutions, hpitch, d_solutions, dpitch,
                                a * sizeof(unsigned int), // width
                                numSolutions,                // height
                                hipMemcpyDeviceToHost));

    printf("Processing time: %f (ms)\n", elapsedTime);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    // print answers
    for (int i = 0; i < numSolutions; i++) {
        unsigned int * solution = &h_solutions[i * n];
        // i is which solution
        printf("Solution %d is [", i);
        for (int k = 0; k < a; k++) {
            printf("%d%s", solution[k], (k < a-1) ? ", " : "]");
        }
        for (int j = 0, k = 0; j < n*n; j++) {
            // j is which square of the chessboard (note each solution
            // must be in sorted order)
            // k is which queen we're placing (also index into the soln. array)
            if (j % n == 0) {
                printf("\n");
            }
            if (j == solution[k]) {
                printf("Q");
                k++;
            } else {
                printf(".");
            }
        }
        printf("\n");
    }

    // clean up memory
    // for 2x2 we don't need to free (h_solutions) at the end
    free(h_solutions);
    CUDA_SAFE_CALL(hipFree(d_solutions));
    hipFree(count);
}
