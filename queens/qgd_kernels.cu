#include "hip/hip_runtime.h"

/* thanks old college classmate Sean Anderson!
 * http://graphics.stanford.edu/~seander/bithacks.html#DetermineIfPowerOf2
 */

 #include "checker_helper.cu"

//  #define BOARDSIZE 5
//  #define NUMQUEENS 3
 
 #define BOARDSIZE 4
 #define NUMQUEENS 2
 
  __global__
  void qgdKernel(int n, int a, bool one, bool all, int pitch,
                 unsigned int * d_solution, unsigned int * count) {
      // this kernel is completely hardcoded to the 4x4 board
      // I'm not pretending otherwise
  
      int tid = threadIdx.x + blockIdx.x * blockDim.x;
  
      // tid is a 16-bit number, where each bit corresponds to 1 square
      // on the chessboard.
      // a 1
  
      // strategy: look at every single possible board in parallel
      // this is very inefficient and scales extremely poorly
      // it is a bad method
      // do not do this to scale to more chessboards
  
      // we know that for n=4, a=2, so knock out all boards where a != 2
      int bitcount = countBits(tid);
      if (bitcount != a) { return; }
  

      int numQueens = NUMQUEENS;
      int boardSize = BOARDSIZE;
      // create queens List for n = 4 the size of the proposed solution is 2
     unsigned int queensList[NUMQUEENS] = {0};
     int temp = tid;
     for (unsigned int c = 0, qi = 0 ; temp ; temp >>= 1, c++){
         if (temp & 1){
             queensList[qi] = c;
             qi++;
         }
     }
    //  if (!checkerFunc (queensList, boardSize, numQueens)) {return;}
 
 
    //   int in_check = 0;           // start with no squares in check
    //   int loc;
    //   for (loc = 0; loc < n*n; loc++) { // iterate over possible queen locations
    //       int queen = 1 << loc;
    //       if (tid & queen) {      // there's a queen at position i
    //           // set the whole row in check
    //           int row = 0xf << (loc & 0xc);
    //           in_check |= row;
    //           // set the whole column in check
    //           int col = 0x1111 << (loc & 0x3);
    //           in_check |= col;
    //           // if we're on a diagonal, set entire diagonal
    //           if (queen & 0x2100) { in_check |= 0x2100; }
    //           if (queen & 0x4210) { in_check |= 0x4210; }
    //           if (queen & 0x8421) { in_check |= 0x8421; }
    //           if (queen & 0x0842) { in_check |= 0x0842; }
    //           if (queen & 0x0084) { in_check |= 0x0084; }
  
    //           if (queen & 0x4800) { in_check |= 0x4800; }
    //           if (queen & 0x2480) { in_check |= 0x2480; }
    //           if (queen & 0x1248) { in_check |= 0x1248; }
    //           if (queen & 0x0124) { in_check |= 0x0124; }
    //           if (queen & 0x0012) { in_check |= 0x0012; }
    //       }
    //   }
    //   if (in_check != 0xffff) { return; }
  
      // if we've reached this point, we have a valid board with configuration tid
      // printf("%x\n", tid);
    //   // claim one of the valid solutions
    //   int solution_id = atomicAdd(count, 1);
  
    //   // the below line sets solution = d_solution[solution_id]
    //   unsigned int* solution =
    //       (unsigned int *) ((char *) d_solution + solution_id * pitch);
  
    //   // solution is of the form [a,b] where a<b and each number
    //   // is an index of a queen into the 1-dimensional n*n-element chessboard
    //   int i, c, k;
    //   for (i = 0, c = tid, k = 0; c != 0; i++, c >>= 1) {
    //       if (c & 1) {
    //           solution[k++] = i;
    //       }
    //   }
    
    if (checkerFunc (queensList, boardSize, numQueens)) 
        addSolution (queensList,  numQueens, d_solution, count, pitch);
  }
  
  // Store your solutions in d_solution, which has already been allocated for you
  void qgd(int n, int a, bool one, bool all, int pitch,
           unsigned long long numSolutions, unsigned int * d_solution,
           unsigned int * count) {
    //   if (one) {
    //       fprintf(stderr, "Instructor's solution only works for -all\n");
    //       exit(42);
    //   }
    //   if (n != 4) {
    //       fprintf(stderr, "Instructor's solution only works for n=4\n");
    //       exit(4);
    //   }
    //   if (a != 2) {
    //       fprintf(stderr, "Instructor's solution only works for a=2\n");
    //       exit(2);
    //   }
  
      // there are 2^16 possible configurations of queens on a 4x4 chessboard
      // 2^8 blocks of 2^8 threads each will check them all (brute force)
      qgdKernel<<< 1<<8, 1<<8 >>>(n, a, one, all, pitch, d_solution, count);
    //   qgdKernel<<< 1<<10, 1<<15 >>>(n, a, one, all, pitch, d_solution, count);
  }
  