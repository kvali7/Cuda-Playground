
#include <hip/hip_runtime.h>

__device__
unsigned long long int combin(int m, int k) {
    unsigned long long int comb = 1;
    for (int j = m; j > m - k; j--) {
        comb = comb * j;
    }
    for (int q = k; q > 0; q--) {
        comb = comb / q;
    }
    return comb;
}

__device__
void queenGen(unsigned int* queensList, unsigned long long int i, int m, int k) {
    //lexico
    int q = 0;
    i  = i + 1;
    int j = 0;
    for (int s = 1; s < k + 1; s++) {
        int cs = j + 1;
        unsigned long long int com = combin(m - cs, k - s);
        while (i > com) {
            i -= com;
            cs += 1;
            com = combin(m - cs, k - s);
        }
        queensList[q++] = cs - 1;
        j = cs;
    }
}

