#include "hip/hip_runtime.h"
#include "comb_creator.cu"
#include "checker_helper.cu"

__global__
void qgdKernelOne(int width, int numQueens, int pitch,
    unsigned int* d_solution, unsigned int* count, unsigned long long int maxit, unsigned int* flag) {

    unsigned long long int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;

    // create queens List of max size 16
        unsigned int queensList[16] = {0};

    for (unsigned long long int id = tid; id < maxit; id += stride){

        if (*flag) return;

        queenGen (queensList, id, width * width, numQueens); // with for loop
        if (!checkerFunc (queensList, width, numQueens) ) {continue;} // with for loop

        // queenGen (queensList, tid, width * width, numQueens); // without for loop
        // if (!checkerFunc (queensList, width, numQueens)) {return;} // without for loop

          addtoSolution (queensList,  numQueens, d_solution, count, pitch);
          int valflag = atomicAdd(flag, 1);
          printf("flag = %d\n",valflag);

    }
  
}

__global__
void qgdKernel(int width, int numQueens, int pitch,
      unsigned int* d_solution, unsigned int* count, unsigned long long int maxit) {


    unsigned long long int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
        // create queens List of max size 16
        unsigned int queensList[16] = {0};

    for (unsigned long long int id = tid; id < maxit; id += stride){
        
        // queenGen (queensList, tid, width * width, numQueens); // without for loop
        // if (!checkerFunc (queensList, width, numQueens)) {return;} // without for loop

        queenGen (queensList, id, width * width, numQueens); // with for loop
        if (!checkerFunc (queensList, width, numQueens)) {continue;} // with for loop


          addtoSolution (queensList,  numQueens, d_solution, count, pitch);

    }
    
}

// Store your solutions in d_solution, which has already been allocated for you
void qgd(int width, int numQueens, bool one, bool all, int pitch,
    unsigned long long numSolutions, unsigned int* d_solution,
    unsigned int* count) {


    unsigned int* flag;
    hipMallocManaged(&flag, sizeof(unsigned int));
    *flag = 0;

    //all combinations
    unsigned long long int comb = 1;
    int m = width * width;
    int k = numQueens;
    for (int j = m; j > m - k; j--) {
        comb = comb * j;
    }
    for (int q = k; q > 0; q--) {
        comb = comb / q;
    }


    unsigned long int blockSize;

    if (width < 7)
        blockSize = 1 << 8;
    else
      //// for RTX 2080 ti and K40 1024 max thread
      // blockSize = 1 << 9;
      ////for titan xp 2048 max thread
      blockSize = 1 << 10;       
      
      
    //dynamic block size for all
    unsigned long int numBlocks;

    // unsigned long long int numBlocksAllTable[] = {
    //   1 << 2, 1 << 2, 1 << 2, 1 << 2, 1 << 2, 1 << 2, 1 << 6, 1 << 8, 1 << 13, 1 << 15, 1 << 16,  1 << 17, 1 << 24, 1 << 27 };

    unsigned long  int numBlocksOneTable[] = {
      1, 1, 1, 1, 1, 1, 50, 8 , 20, 700, 2000, 2000,  2000 , 2000, 2000, 2000, 2000, 2000, 2000, 2000, 2000, 2000 };



    unsigned long long int maxit = comb;
    if (all == true){
        numBlocks = comb / blockSize + 4;
    }
    else{
        numBlocks = numBlocksOneTable[width];
        if (width == 12)
          maxit = 309000000;
    }
   
    // if (numBlocks < 1)
    //   numBlocks = 1;
    // if (all == true){


// titan xp
    //      n = 2   0.32
    //  n = 3   0.32
    //  n = 4   0.37
    //  n = 5   0.5 ms
    //   n = 6   0.6 ms
    //     n = 7   4 ms
    // n = 8   232 ms
    // n = 9   756 ms
    //n = 10   1 secs
    //n = 11    1.2 secs
    // n = 12   70 secs 
    // n = 13  273 secs

    // if (one == true){
    // n = 2   0.27
    //  n = 3   0.32
    //   n = 4   0.33
    //  n = 5   0.4 ms
    // n = 6   0.8 ms
    //  n = 7   0.8 ms
    //   n = 8   1.5 ms
    //   n = 9   80 ms
    //  n = 10   365 ms
    //   n = 11    856 ms
    //     n = 12   1.6 secs 
    //   n = 13     4 secs
    //  n = 14     325 secs

    //// in k40 240 = 16* 15 SMs
    // if (numBlocks > 480)
    //   numBlocks = 480;
    //// in titan xp = 32* 30 SMs
    if (numBlocks > 2000)
      numBlocks = 2000;
    //// in rtx 2080 ti = 16* 68 SMs
    // if (numBlocks > 2200)
    //   numBlocks = 2200;

    // blockSize = 256;
    // numBlocks = 4;


    // maxit = 7000;
    // blockSize = 256;
    // numBlocks = 10;

    printf("The number of the total combinations is %llu\n", comb);
     unsigned long long int threads;
    threads = numBlocks * blockSize;
    printf("maxit = %llu, thread = %d, blocks = %d The number of the total threads is %llu\n", maxit, blockSize, numBlocks, threads);



    if (all == true){

        qgdKernel <<< numBlocks, blockSize >>> (width, numQueens, pitch, d_solution, count, maxit);
    }
    else {

        qgdKernelOne <<< numBlocks, blockSize >> > (width, numQueens, pitch, d_solution, count, maxit, flag);
    }

}
