#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include "checker_helper.cu"

#define BLOCK_SIZE 4
#define NUM_BLOCKS 4
#define GLOBAL_BUF_SIZE 16384
#define INTERNAL_BUF_SIZE BLOCK_SIZE

__device__ volatile int globalHead;
__device__ volatile int globalTail;
__device__ volatile int blockFence;
// __device__ volatile int tailSem = 0;
// __device__ volatile int headSem = 0;

// __device__ void acquire_semaphore(volatile int *lock){
//     while (atomicCAS((int *) lock, 0, 1) != 0);
// }

// __device__ void release_semaphore(volatile int *lock){
//     *lock = 0;
//     __threadfence();
// }


__global__ void initialize(int *buffer, int width, size_t pitch){
    globalHead = 0;
    globalTail = 0;
    blockFence = 0;
    int numQueens = 5;
    for (int i = 0; i < ceilf(width / 2); ++i)
        for (int j = 0; j < ceilf(width / 2); ++j){
            int *row = (int*) ((char*) buffer + (globalTail++) * pitch);
            row[0] = i * width + j;
            // printf("globalTail: %4d\telement: %4d\n", globalTail, row[0]);
        }
    for (int i = 0; i < width * width; ++i){
        int *row = (int*) ((char*) buffer + i * pitch);
        for (int q = 0; q < numQueens; ++q){
            printf("%4d\t", row[q]);
        }
        printf("\n");
        row[0] = 2;
        row[1] = 20;
        row[2] = 11;
        row[3] = 30;
        row[4] = 60;
        if (!checkerFunc (row, width, numQueens)) {printf("this is Not a Solution!\n");}
        row[0] = 27;
        row[1] = 33;
        row[2] = 56;
        row[3] = 4;
        row[4] = 55;
        if (!checkerFunc (row, width, numQueens)) {printf("s2 is Not a Solution!\n");}
    }
    
}
 
__global__ void qgdKernel(int width, size_t pitch, int numQueens, int *globalBuffer){

    int globalIndex = threadIdx.x + blockIdx.x * blockDim.x;
    int internalIndex = threadIdx.x; 
    int blockIndex = blockIdx.x;

    __shared__ int internalBuffer[16][INTERNAL_BUF_SIZE];
    __shared__ int internalHead;
    __shared__ int internalTail;

    if (internalIndex == 0){
        internalHead = 0;
        internalTail = 0;
    }
    __syncthreads();

    for (int i = 0; i < 16; ++i)
        internalBuffer[i][internalIndex] = -1;

    __syncthreads();
    while(globalHead < globalTail){
        if (internalIndex == 0){
            while(blockFence != blockIndex);
            while(true){
                if (internalTail == INTERNAL_BUF_SIZE)
                    break;
                if (globalHead == globalTail)
                    break;
                
                int *row = (int *)((char *) globalBuffer + globalHead * pitch);
                for (int i = 0; i < numQueens; ++i){
                    internalBuffer[i][internalTail] = row[i];
                    row[i] = -1;
                }
                ++internalTail;
                ++globalHead;
                if (globalHead == globalTail){
                    globalHead = 0;
                    globalTail = 0;
                }
            }
            if (++blockFence == NUM_BLOCKS)
                blockFence = 0;
        }
        __syncthreads();
        if (internalIndex == 0){
            while(blockFence != blockIndex);
            while(true){
                if (globalTail == GLOBAL_BUF_SIZE)
                    break;
                if (internalHead == internalTail){
                    internalHead = 0;
                    internalTail = 0;
                    break;
                }
                
                int *row = (int *)((char *) globalBuffer + globalTail * pitch);
                for (int i = 0; i < numQueens; ++i){
                    row[i] = internalBuffer[i][internalHead];
                    internalBuffer[i][internalHead] = -1;
                }
                ++internalHead;
                ++globalTail;
                if (internalHead == internalTail){
                    internalHead = 0;
                    internalTail = 0;
                }
            }
            if (++blockFence == NUM_BLOCKS)
                blockFence = 0;
        }
        __syncthreads();

        break;
    }
}

int main(void){
    int width = 8;
    int numQueens = 5;

    int *buffer;
    size_t pitch;

    hipMallocPitch((void**) &buffer, &pitch, numQueens * sizeof(int), width * width);
    hipMemset2D(buffer, pitch, 255, numQueens * sizeof(int), width * width);
    
    initialize<<< 1, 1 >>>(buffer, width, pitch);
    hipDeviceSynchronize();
    qgdKernel<<< 4, 4 >>>(width, pitch, numQueens, buffer);
    hipDeviceSynchronize();

    hipFree(buffer);
}
 
 