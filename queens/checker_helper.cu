
#include <hip/hip_runtime.h>


__device__
int countBits(unsigned int v) {
    int c; // c accumulates the total bits set in v
    for (c = 0; v; v >>= 1) {
        c += v & 1;
    }
    return c;
}

__device__
bool checkerFunc (int* queensList,int width, int numQueens){
    ////input exceptions
    // if (numQueens > width){
    //     printf("The Number of Queens is greater than width of the board\n");
    // }
    bool ifCheck = true;
    ////max we can do is 32 x 32
    unsigned int in_checkArr[32] ={0};
    for (int queen = 0; queen < numQueens; queen++){
        int posqueen = queensList[queen];
        // if (posqueen < 0 || posqueen >= width * width){
        //     printf("The position  of Queen is invalid\n");
        // }
        int row = posqueen/width;
        int col = posqueen % width;
        ////row easy!
        in_checkArr[row] |=  0xffffffff;
        for (int r = 0; r < width; r++){
            ////column in loop
            in_checkArr[r] |=  1 << col;
            ////main diagon
            if (row + col - r < width && row + col -r >= 0) 
                in_checkArr[r] |=  1 << row + col - r;
            ////other diagon
            if (col - row + r >= 0 && col - row + r < width) 
                in_checkArr[r] |=1 << col - row + r;
            if (countBits(in_checkArr[r]) < width && queen == numQueens - 1)
                ifCheck = false;
        }
    }

    return ifCheck; 
}

__device__
unsigned int* addSolution (int tid, unsigned int* d_solution, unsigned int* count, int pitch){
    // claim one of the valid solutions
    int solution_id = atomicAdd(count, 1);

    // the below line sets solution = d_solution[solution_id]
    unsigned int* solution =
        (unsigned int*) ((char*) d_solution + solution_id * pitch);

    // solution is of the form [a,b] where a<b and each number
    // is an index of a queen into the 1-dimensional n*n-element chessboard
    int i, c, k;
    for (i = 0, c = tid, k = 0; c != 0; i++, c >>= 1) {
        if (c & 1) {
            solution[k++] = i;
        }
    }
    return solution;
}