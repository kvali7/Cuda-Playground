
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>
#include <math.h>

using namespace std; 

__global__
void p_vec_dist(int dim, float3 p, float3 *vec, float *res){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < dim; i += stride){
        res[i] = (p.x - vec[i].x) * (p.x - vec[i].x);   
        res[i] += (p.y - vec[i].y) * (p.y - vec[i].y);
        res[i] += (p.z - vec[i].z) * (p.z - vec[i].z);
    }
}

__global__
void vec_vec_dist(int dim, float3 *vec0, float3 *vec1, float *res){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < dim; i += stride){
        //p_vec_dist<<<1, dim>>>(dim, vec0[i], vec1, res[i]);
        for (int j = 0; j < dim; j++){
            res[i] = (vec0[i].x - vec1[j].x) * (vec0[i].x - vec1[j].x);
            res[i] += (vec0[i].y - vec1[j].y) * (vec0[i].y - vec1[j].y);
            res[i] += (vec0[i].z - vec1[j].z) * (vec0[i].z - vec1[j].z);   
        }
    }
}

__global__
void fill_float3(int dim, float3 val, float3 *dst)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int i = index; i < dim; i += stride){
    dst[i].x = val.x;
    dst[i].y = val.y;
    dst[i].z = val.z;
  }
}

int main(void){
    int dim = 1 << 10;
    float3 *x, *y; 
    float *res; 

    hipMallocManaged(&x, dim * sizeof(float3));
    hipMallocManaged(&y, dim * sizeof(float3));
    hipMallocManaged(&res, dim * sizeof(float));

    fill_float3<<<32, 32>>>(dim, make_float3(1.0, 2.0, 3.0), x);
    fill_float3<<<32, 32>>>(dim, make_float3(4.0, 5.0, 6.0), y);

    vec_vec_dist<<<32, 32>>>(dim, x, y, res);

    float maxError = 0.0;
    for (int i = 0; i < dim; i++)
        for (int j = 0; j < dim; j++)
            maxError = fmax(maxError, fabs(res[i] - 0.0f));
    cout << "Max error: " << maxError << endl;

    hipFree(x);
    hipFree(y);
    hipFree(res);
    return 0;
}