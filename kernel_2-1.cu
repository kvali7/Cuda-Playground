
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>
#include <math.h>

/*
dst0 = src + 1;

dst1 = dst0 + 1;
dst1 = src + 2;

dst2 = dst1 + 1;
dst2 = dst0 + 2;
dst2 = src + 3;
*/

// Kernel function to copy the elements  
// of one array to two more arrays.
__global__
void cpy_float4(int n, float4 *src, float4 *dst0, float4 *dst1, float4 *dst2)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int i = index; i < n; i += stride){
    if (index > 0){
      dst0[i].x = src[i].x + src[i - 1].x;
      dst0[i].y = src[i].y + src[i - 1].y;
      dst0[i].z = src[i].z + src[i - 1].z;
      dst0[i].w = src[i].w + src[i - 1].w;
      dst1[i].x = dst0[i].x + dst0[i - 1].x;
      dst1[i].y = dst0[i].y + dst0[i - 1].y;
      dst1[i].z = dst0[i].z + dst0[i - 1].z;
      dst1[i].w = dst0[i].w + dst0[i - 1].w;
      dst2[i].x = dst1[i].x + dst1[i - 1].x;
      dst2[i].y = dst1[i].y + dst1[i - 1].y;
      dst2[i].z = dst1[i].z + dst1[i - 1].z;
      dst2[i].w = dst1[i].w + dst1[i - 1].w;
    }
    else {
      dst0[i].x = src[i].x + 0; 
      dst0[i].y = src[i].y + 0; 
      dst0[i].z = src[i].z + 0; 
      dst0[i].w = src[i].w + 0; 
      dst1[i].x = dst0[i].x + 0;
      dst1[i].y = dst0[i].y + 0;
      dst1[i].z = dst0[i].z + 0;
      dst1[i].w = dst0[i].w + 0;
      dst2[i].x = dst1[i].x + 0;
      dst2[i].y = dst1[i].y + 0;
      dst2[i].z = dst1[i].z + 0;
      dst2[i].w = dst1[i].w + 0;  
    }
  }
}

__global__
void fill_float4(int n, float4 *dst)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int i = index; i < n; i += stride){
    dst[i].x = 0;
    dst[i].y = 0;
    dst[i].z = 0;
    dst[i].w = 0;
  }
}

int main(void)
{
  int N = 1 << 20;
  float4 *x, *y, *z, *w;
  
  // Allocate Unified Memory – accessible from CPU or GPU
  hipMallocManaged(&x, N * sizeof(float4));
  hipMallocManaged(&y, N * sizeof(float4));
  hipMallocManaged(&z, N * sizeof(float4));
  hipMallocManaged(&w, N * sizeof(float4));

  int blockSize = 256;
  int numBlocks = (N + blockSize - 1) / blockSize;

  
  // initialize x  
  fill_float4<<<numBlocks, blockSize>>>(N, x);
  
  // Run kernel on 1M elements on the GPU
  cpy_float4<<<numBlocks, blockSize>>>(N, x, y, z, w);

  // Wait for GPU to finish before accessing on host
  hipDeviceSynchronize();

  double maxError0 = 0.0;
  double maxError1 = 0.0;
  double maxError2 = 0.0;
  double maxError3 = 0.0;

  for (int i = 0; i < N; i++){
    maxError0 = fmax(maxError0, fabs(y[i].x - 1.0));
    maxError1 = fmax(maxError1, fabs(y[i].y - 1.0));
    maxError2 = fmax(maxError2, fabs(y[i].z - 1.0));
    maxError3 = fmax(maxError3, fabs(y[i].w - 1.0));

    maxError0 = fmax(maxError0, fabs(z[i].x - 2.0));
    maxError1 = fmax(maxError1, fabs(z[i].y - 2.0));
    maxError2 = fmax(maxError2, fabs(z[i].z - 2.0));
    maxError3 = fmax(maxError3, fabs(z[i].w - 2.0));
    
    maxError0 = fmax(maxError0, fabs(w[i].x - 3.0));
    maxError1 = fmax(maxError1, fabs(w[i].y - 3.0));
    maxError2 = fmax(maxError2, fabs(w[i].z - 3.0));
    maxError3 = fmax(maxError3, fabs(w[i].w - 3.0));
  }
  std::cout << "Max error0: " << maxError0 << std::endl;
  std::cout << "Max error1: " << maxError1 << std::endl;
  std::cout << "Max error2: " << maxError2 << std::endl;
  std::cout << "Max error3: " << maxError3 << std::endl;

  // Free memory
  hipFree(x);
  hipFree(y);
  hipFree(z);
  hipFree(w);

  return 0;
}
