#include "hip/hip_runtime.h"
#include "comb_creator.cu"
#include "checker_helper.cu"

__global__
void qgdKernelOne(int width, int numQueens, int pitch,
    unsigned int* d_solution, unsigned int* count, unsigned long long int minit, unsigned long long int maxit, unsigned int* flag) {

    unsigned long long int tid = threadIdx.x + blockIdx.x * blockDim.x + minit;
    int stride = blockDim.x * gridDim.x;

    // create queens List of max size 16
        unsigned int queensList[16] = {0};

    for (unsigned long long int id = tid; id < maxit ; id += stride){

        if (*flag) return;

        queenGen (queensList, id, width * width, numQueens); // with for loop
        if (!checkerFunc (queensList, width, numQueens) ) {continue;} // with for loop

        // queenGen (queensList, tid, width * width, numQueens); // without for loop
        // if (!checkerFunc (queensList, width, numQueens)) {return;} // without for loop

          addtoSolution (queensList,  numQueens, d_solution, count, pitch);
          int valflag = atomicAdd(flag, 1);
          printf("id = %llu, tid = %llu, flag = %d\n",id, tid, valflag);
        // printf("count = %u\n",*count);

    }
  
}

// max loops 20096001

__global__
void qgdKernel(int width, int numQueens, int pitch,
      unsigned int* d_solution, unsigned int* count, unsigned long long int minit, unsigned long long int maxit) {


    unsigned long long int tid = threadIdx.x + blockIdx.x * blockDim.x + minit;
    int stride = blockDim.x * gridDim.x;
        // create queens List of max size 16
        unsigned int queensList[16] = {0};

    for (unsigned long long int id = tid; id < maxit; id += stride){
        // queenGen (queensList, tid, width * width, numQueens); // without for loop
        // if (!checkerFunc (queensList, width, numQueens)) {return;} // without for loop

        queenGen (queensList, id, width * width, numQueens); // with for loop
        if (!checkerFunc (queensList, width, numQueens)) {continue;} // with for loop
          addtoSolution (queensList,  numQueens, d_solution, count, pitch);
        // printf("count = %u\n",*count);
    }
    
}

// Store your solutions in d_solution, which has already been allocated for you
void qgd(int width, int numQueens, bool one, bool all, int pitch,
    unsigned long long numSolutions, unsigned int* d_solution,
    unsigned int* count) {


    unsigned int* flag;
    hipMallocManaged(&flag, sizeof(unsigned int));
    *flag = 0;

    //all combinations
    unsigned long long int comb = 1;
    int m = width * width;
    int k = numQueens;
    for (int j = m; j > m - k; j--) {
        comb = comb * j;
    }
    for (int q = k; q > 0; q--) {
        comb = comb / q;
    }


    unsigned long int blockSize;

    if (width < 7)
        blockSize = 1 << 8;
    else
      //// for RTX 2080 ti and K40 1024 max thread
      // blockSize = 1 << 9;
      ////for titan xp 2048 max thread
      blockSize = 1 << 10;       
      
      
    //dynamic block size for all
    unsigned long int numBlocks;


    unsigned long  int numBlocksOneTable[] = {
      1, 1, 1, 1, 1, 1, 50, 8 , 20, 700, 2000, 2000,  2000 , 2000, 2000, 2000, 2000, 2000, 2000, 2000, 2000, 2000 
    };



    unsigned long long int maxit = comb;
    unsigned long long int minit = 0;
    unsigned long long int tempmaxit;
    unsigned long long int initialmaxit;

    if (all == true){
        numBlocks = comb / blockSize + 4;
    }
    else{
        numBlocks = numBlocksOneTable[width];
        if (width == 12)
          maxit = 309000000;
        if (width == 13)
          maxit = 30009000000;
    }
    tempmaxit = maxit;
    initialmaxit = maxit;
    // if (numBlocks < 1)
    //   numBlocks = 1;
    // if (all == true){


// titan xp
    //      n = 2   0.32
    //  n = 3   0.26 ms
    //  n = 4   0.34 ms
    //  n = 5   0.48 ms
    //   n = 6   0.52 ms
    //     n = 7   5 ms
    // n = 8   240 ms
    // n = 9   530 ms
    //n = 10   800 ms
    //n = 11    1.3 secs
    // n = 12   70 secs 
    // n = 13  273 secs

    // if (one == true){
    // n = 2   0.27 ms
    //  n = 3   0.27 ms
    //   n = 4   0.33 ms
    //  n = 5   0.6 ms
    // n = 6   0.6 ms
    //  n = 7   1.4 ms
    //   n = 8   1 ms
    //   n = 9   5 ms
    //  n = 10   119 ms
    //   n = 11    750 ms
    //     n = 12   1.6 secs 
    //   n = 13     4 secs
    //  n = 14     325 secs


    //// in k40 240 = 16* 15 SMs
    // if (numBlocks > 480)
    //   numBlocks = 480;
    //// in titan xp = 32* 30 SMs
    if (numBlocks > 2000)
      numBlocks = 2000;
    //// in rtx 2080 ti = 16* 68 SMs
    // if (numBlocks > 2200)
    //   numBlocks = 2200;


    // printf("The number of the total combinations is %llu\n", comb);
    //  unsigned long long int threads;
    // threads = numBlocks * blockSize;
    // printf("maxit = %llu, thread = %d, blocks = %d The number of the total threads is %llu\n", maxit, blockSize, numBlocks, threads);


    unsigned int tempcount;
    tempcount = *count;
    unsigned long long int range = 100000000;

    if (all == true){
        if ((tempmaxit - minit) >= range)
            for (maxit = range + minit; (tempmaxit - minit) >= range; minit = maxit, maxit += range){
                qgdKernel <<< numBlocks, blockSize >>> (width, numQueens, pitch, d_solution, count, minit, maxit);
                hipDeviceSynchronize();
                printf("count = %llu so we found %llu solutions in the iteration from minit = %llu to maxit = %llu\n",*count, *count - tempcount, minit,maxit);
                tempcount = *count;
            }
        qgdKernel <<< numBlocks, blockSize >>> (width, numQueens, pitch, d_solution, count, minit, initialmaxit);
        hipDeviceSynchronize();
        printf("count = %u so we found %llu solutions in the iteration from minit = %llu to maxit = %llu\n",*count, *count - tempcount, minit, initialmaxit);
        
    }
    else {

        qgdKernelOne <<< numBlocks, blockSize >> > (width, numQueens, pitch, d_solution, count, minit, maxit, flag);
    }

}
