

#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>

__device__
int combin(int m, int k ){
    unsigned long int comb = 1;
    if (m > 0 && k > 0 ) {
        for (int j = m ; j > m - k; j--){
            comb = comb * j;
        }
        for (int q = k; q > 0; q--){
            comb = comb / q;
        }
    }
    return comb;
}

__device__
void queenGen(unsigned int* queensList, int i, int m, int k){
    int q = 0;
    int r = i +1;
    int j = 0;
    for (int s = 1; s < k + 1; s++){
        int cs = j+1;
        int com = combin(m-cs,k-s);
        while ((r - com)>0){
            r -= com;
            cs += 1;
            com = combin(m-cs,k-s);
        }
        queensList[q++] = cs-1;
        j = cs;
    }
}
