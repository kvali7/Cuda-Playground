#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>
//#include <hip/hip_runtime.h>

// function to add the elements of two arrays
__global__
void add(int n, float *x, float *y)
{
  //int index = 0;
  //int stride = 1;
  //int index = threadIdx.x;
  //int stride = blockDim.x;
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int i = index; i < n; i += stride)
      y[i] = x[i] + y[i];
}

//__global__
//void init(int n, float* x, float* y){
//
//  int index = blockIdx.x * blockDim.x + threadIdx.x;
//  int stride = blockDim.x * gridDim.x;
//  for (int i = index; i < n; i += stride){
//     x[i] = 1.0f;
//     y[i] = 2.0f;
//  }
//}

int main(void)
{
  int N = 1<<20; // 1M elements

  int blockSize = 256;
  int numBlocks = (N + blockSize - 1) / blockSize;

  //variable defition on system memory
  //float *x = new float[N];
  //float *y = new float[N];
 
  float *x, *y;
  //variable allocation on GPU memory
  hipMallocManaged (&x, N*sizeof(float));
  hipMallocManaged (&y, N* sizeof(float));



  
  // initialize x and y arrays on the host
  //for (int i = 0; i < N; i++) {
  //  x[i] = 1.0f;
  //  y[i] = 2.0f;
  //}
  //init<<<numBlocks, blockSize>>>(N, x, y);


  // Prefetch the data to the GPU
  int device = -1;
  hipGetDevice(&device);
  hipMemPrefetchAsync(x, N*sizeof(float), device, NULL);
  hipMemPrefetchAsync(y, N*sizeof(float), device, NULL);

  // Run kernel on 1M elements on the CPU
  //add(N, x, y);
  
  // Run kernel on 1M parallel elements on the GPU  
  //add<<<1,256>>>(N, x, y);
  // Multiple blocks
  add<<<numBlocks, blockSize>>>(N, x, y);

  // wait for the GPU to finish the results
  hipDeviceSynchronize();

  // Check for errors (all values should be 3.0f)
  float maxError = 0.0f;
  for (int i = 0; i < N; i++)
    maxError = fmax(maxError, fabs(y[i]-3.0f));
  std::cout << "Max error: " << maxError << std::endl;

  // Free memory
  //delete [] x;
  //delete [] y;
  
  // Free GPU memory
  hipFree(x);
  hipFree(y);


  
  return 0;
}
